#include "hip/hip_runtime.h"
// @file multiboxdetector_gpu.cu
// @brief Multibox Detector GPU implementation, 
// based on Wei Liu's SSD caffe code
// @author Samuel Albanie
// @author Andrea Vedaldi

/*
Copyright (C) 2017- Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "multiboxdetector.hpp"
#include <bits/data.hpp>
#include <assert.h>
#include <float.h>
#include <cstdio>
#include <algorithm>
#include <math.h>
#include <string.h>
#include <map>
#include <vector>

/* ------------------------------------------------------------ */
/*                                                      kernels */
/* ------------------------------------------------------------ */

enum {
  XMIN = 0,
  YMIN,
  XMAX,
  YMAX,
} ;


template <typename T>
__global__ void decodeBoxesKernel(const int numThreads,
                                  const int numPriors, 
                                  const T* locPreds, 
                                  const T* priors,
                                  T* decodedBoxes) 
{
    // Grid stride-loop 
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; 
             index < numThreads ; 
             index += blockDim.x * gridDim.x) 
    {
        const int modIndex = index % 4 ;
        const int priorIndex = ((index / 4 ) % numPriors) * 4 ;
        const int varIndex = priorIndex + numPriors * 4 ;

        if ((modIndex == XMIN) || (modIndex == XMAX)) {

            const T priorXmin = priors[priorIndex] ;
            const T priorXmax = priors[priorIndex + 2] ;
            const T priorWidth = priorXmax - priorXmin;
            const T priorCenterX = (priorXmin + priorXmax) / 2.;
            const T xmin = locPreds[index - modIndex];
            const T xmax = locPreds[index - modIndex + 2];

            T decodedWidth ; 
            T decodedCenterX ; 

            decodedCenterX = priors[varIndex] * xmin * priorWidth + priorCenterX ;
            decodedWidth = exp(priors[varIndex + 2] * xmax) * priorWidth ;

            if (modIndex == XMIN) {
                decodedBoxes[index] = decodedCenterX - decodedWidth / 2.;
            } else {
                decodedBoxes[index] = decodedCenterX + decodedWidth / 2.;
            }

        } else {

            const T priorYmin = priors[priorIndex + 1] ;
            const T priorYmax = priors[priorIndex + 3] ;
            const T priorHeight = priorYmax - priorYmin;
            const T priorCenterY = (priorYmin + priorYmax) / 2.;
            const T ymin = locPreds[index - modIndex + 1];
            const T ymax = locPreds[index - modIndex + 3];

            T decodedHeight ;
            T decodedCenterY ;

            decodedCenterY = priors[varIndex + 1] * ymin * priorHeight + priorCenterY ;
            decodedHeight = exp(priors[varIndex + 3] * ymax) * priorHeight ;

            if (modIndex == YMIN) {
                decodedBoxes[index] = decodedCenterY - decodedHeight / 2.;
            } else {
                decodedBoxes[index] = decodedCenterY + decodedHeight / 2.;
            }
        }
    }
}

template <typename T>
__global__ void permuteConfsKernel(const int numThreads,
                                   const int numClasses, 
                                   const int numPriors,
                                   const T* confPreds, 
                                   T* permuted) 
{
    // Grid stride-loop 
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; 
             index < numThreads ; 
             index += blockDim.x * gridDim.x) 
    {
        const int classIndex = index % numClasses;
        const int priorIndex = (index / numClasses) % numPriors;
        const int batchIndex = (index / numClasses) / numPriors;
        const int newIndex = (batchIndex * numClasses + classIndex) * numPriors + priorIndex ;
        permuted[newIndex] = confPreds[index];
    }
}

/* ------------------------------------------------------------ */
/*                                              kernel wrappers */
/* ------------------------------------------------------------ */

template <typename T>
void decodeBoxesGPU(const int numThreads,
                    const int numPriors, 
                    const T* locPreds, 
                    const T* priors,
                    T* decodedBoxes) 
{
    int numBlocks = (numThreads + 511) / 512 ;
    decodeBoxesKernel<T><<<numBlocks, 512>>>(numThreads,  
                                             numPriors, 
                                             locPreds, 
                                             priors, 
                                             decodedBoxes) ;
    hipError_t status = hipPeekAtLastError() ;
    if (status != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(status));
        exit(-1) ;
    }
}

template <typename T>
void permuteConfsGPU(const int numThreads,
                    const int numClasses, 
                    const int numConfPreds,
                    const T* confPreds, 
                    T* permuted) 
{
    int numBlocks = (numThreads + 511) / 512 ;
    permuteConfsKernel<T><<<numBlocks, 512>>>(numThreads, 
                                              numClasses, 
                                              numConfPreds,
                                              confPreds, 
                                              permuted);
    hipError_t status = hipPeekAtLastError() ;
    if (status != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(status));
        exit(-1) ;
    }
}  

/* ------------------------------------------------------------ */
/*                                             non kernel utils */
/* ------------------------------------------------------------ */

template <typename T>
bool sortScorePairDescend(const std::pair<float, T>& pairA,
                          const std::pair<float, T>& pairB) 
{
    return pairA.first > pairB.first ;
}

template <typename T>
T getBoxSize(const T* box) {
    T boxSize ;
    if (box[2] < box[0] || box[3] < box[1]) {
        // return 0 for invalid boxes
        boxSize = T(0.) ;
    } else {
        const T width = box[2] - box[0];
        const T height = box[3] - box[1];
        boxSize = width * height;
    }
    return boxSize ;
}

template <typename T>
T jaccardOverlap(const T* boxA, const T* boxB) 
{
    if (boxB[0] > boxA[2] || 
        boxB[2] < boxA[0] ||
        boxB[1] > boxA[3] || 
        boxB[3] < boxA[1]) {
        // return 0 for invalid boxes
        return T(0.) ;
    } else {
        const T xminIntersection = std::max(boxA[0], boxB[0]) ;
        const T yminIntersection = std::max(boxA[1], boxB[1]) ;
        const T xmaxIntersection = std::min(boxA[2], boxB[2]) ;
        const T ymaxIntersection = std::min(boxA[3], boxB[3]) ;

        const T widthIntersection = xmaxIntersection - xminIntersection;
        const T heightIntersection = ymaxIntersection - yminIntersection;
        const T sizeIntersection = widthIntersection * heightIntersection;

        const T sizeBoxA = getBoxSize(boxA);
        const T sizeBoxB = getBoxSize(boxB);

        return sizeIntersection / (sizeBoxA + sizeBoxB - sizeIntersection);
    }
}

template <typename T>
void getMaxScoreIndexCPU(const T* scores, 
                         const float thresh,
                         const int numPriors,
                         const int topK, 
                         std::vector<std::pair<float, int> > *scoreIndexPairs) 
{
    // generate index score pairs for sufficiently high scores
    for (int i = 0 ; i < numPriors ; ++i) {
        if (scores[i] > thresh) {
            scoreIndexPairs->push_back(std::make_pair(scores[i], i)) ;
        }
    }

    // sort the score pair according to the scores in descending order
    std::stable_sort(scoreIndexPairs->begin(), scoreIndexPairs->end(), 
                                       sortScorePairDescend<int>) ;

    // Keep top k scores if needed.
    if (topK > -1 && topK < scoreIndexPairs->size()) {
      scoreIndexPairs->resize(topK) ;
    }
}

template <typename T>
void applyFastNMSCPU(const T* boxes,
                     const T* scores, 
                     const float confThresh,
                     const float nmsThresh, 
                     const int numPriors,
                     const int keepTopK,
                     std::vector<int> *indices) 
{
    // retrieve top k scores (with corresponding indices).
    std::vector<std::pair<float, int> > scoreIndexPairs ;
    getMaxScoreIndexCPU(scores, confThresh, numPriors, keepTopK, &scoreIndexPairs) ;

    // run the nms - note we don't use adaptive NMS here
    int eta = 1 ;
    float adaptiveThresh = nmsThresh ;
    indices->clear() ;

    while (scoreIndexPairs.size() != 0) {
        const int idx = scoreIndexPairs.front().second ;
        bool keep = true ;
        int sz = indices->size() ;
        for (int k = 0 ; k < indices->size() ; ++k) {
            if (keep) {
                const int keptIdx = (*indices)[k] ;
                float overlap = jaccardOverlap(boxes + idx * 4, 
                                               boxes + keptIdx * 4) ;
                keep = overlap <= adaptiveThresh ;
            } else {
                break ;
            }
        }
        if (keep) {
            indices->push_back(idx) ;
        }
        scoreIndexPairs.erase(scoreIndexPairs.begin()) ;
        if (keep && eta < 1 && adaptiveThresh > 0.5) {
            adaptiveThresh *= eta ;
        }
    }
}


/* ------------------------------------------------------------ */
/*                                                      forward */
/* ------------------------------------------------------------ */

namespace vl { namespace impl {

    template<typename T>
    struct multiboxdetector<vl::VLDT_GPU,T>
    {

    static vl::ErrorCode
    forward(Context& context,
            T* output,
            T const* locPreds,
            T const* confPreds,
            T const* priors,
            int nmsTopK,
            int keepTopK,
            int numClasses,
            float nmsThresh,
            float confThresh, 
            int backgroundLabel, 
            size_t outHeight, 
            size_t outWidth, 
            size_t batchSize, 
            size_t numPriors) 
{
    // The first two steps of the forward pass are performed on the GPU i.e.
    //
    // 1. Decoding the box predictions
    // 2. Permuting the confidence scores
    //
    // Following this, the data is returned to the CPU and the NMS is run 
    // serially - this can be updated when we have time :)

    const int BOXES_ARRAY_SIZE = numPriors * 4 * batchSize ;
    const int BOXES_ARRAY_BYTES = BOXES_ARRAY_SIZE * sizeof(T) ;
    T * decodedBoxes ;
    hipMalloc( (void **) &decodedBoxes, BOXES_ARRAY_BYTES) ;

    const int numLocPreds = numPriors * 4 * batchSize ;
    decodeBoxesGPU<T>(numLocPreds, 
                      numPriors, 
                      locPreds, 
                      priors, 
                      decodedBoxes) ;

    // permute the confidence predictions to allow contiguous access
    const int CONF_ARRAY_SIZE = numPriors * numClasses * batchSize ;
    const int CONF_ARRAY_BYTES = CONF_ARRAY_SIZE * sizeof(T) ;
    T * permutedConfPreds ;
    hipMalloc( (void **) &permutedConfPreds, CONF_ARRAY_BYTES) ;

    const int numConfPreds = numPriors * numClasses * batchSize ;
    permuteConfsGPU<T>(numConfPreds, 
                       numClasses, 
                       numPriors, 
                       confPreds,
                       permutedConfPreds) ;

    // allocate on host, copy data back and free up the GPU
    T* h_decodedBoxes = new T [BOXES_ARRAY_SIZE] ;
    T*  h_permutedConfPreds = new T [CONF_ARRAY_SIZE] ;

    hipMemcpy(h_decodedBoxes, decodedBoxes, 
               BOXES_ARRAY_BYTES, hipMemcpyDeviceToHost) ;
    hipMemcpy(h_permutedConfPreds, permutedConfPreds, 
               CONF_ARRAY_BYTES, hipMemcpyDeviceToHost) ;
    hipFree(decodedBoxes) ;
    hipFree(permutedConfPreds) ;

    int numKept = 0 ;
    std::vector<std::map<int, std::vector<int> > > batchIndices ;

    for (int i = 0; i < batchSize; ++i) {

        int numDetections = 0 ;
        std::map<int, std::vector<int> > indices ;
        int confIdxOffset = numClasses * numPriors * i ;
        int boxIdxOffset = numPriors * 4 * i ;

        for (int c = 0 ; c < numClasses ; ++c) {
          if ((c + 1) == backgroundLabel) { // ignore background (MATLAB indexing)
            continue ;
          }
          T* boxes_ = h_decodedBoxes + boxIdxOffset;
          T* confPreds_ = h_permutedConfPreds + confIdxOffset + c * numPriors ;


          applyFastNMSCPU(boxes_, 
                          confPreds_, 
                          confThresh, 
                          nmsThresh, 
                          numPriors, 
                          nmsTopK, 
                          &(indices[c])) ;
          numDetections += indices[c].size() ;
        }

        if (keepTopK > -1 && numDetections > keepTopK) {
            std::vector<std::pair<float, std::pair<int, int> > > scoreIndexPairs ;
            for (std::map<int, std::vector<int> >::iterator it = indices.begin() ;
                 it != indices.end(); ++it) {
                int label = it->first ;
                const std::vector<int>& labelIndices = it->second ;
                for (int j = 0; j < labelIndices.size(); ++j) {
                  int idx = labelIndices[j] ;
                  float score = h_permutedConfPreds[confIdxOffset + label * numPriors + idx] ;
                  scoreIndexPairs.push_back(std::make_pair(
                                         score, std::make_pair(label, idx))) ;
                }
            }

            // Keep top k results per image.
            std::sort(scoreIndexPairs.begin(), scoreIndexPairs.end(),
                      sortScorePairDescend<std::pair<int, int> >);
            scoreIndexPairs.resize(keepTopK);

            // Store the new indices.
            std::map<int, std::vector<int> > newIndices;
            for (int j = 0; j < scoreIndexPairs.size(); ++j) {
                int label = scoreIndexPairs[j].second.first;
                int idx = scoreIndexPairs[j].second.second;
                newIndices[label].push_back(idx);
            }
            batchIndices.push_back(newIndices);
            numKept += keepTopK;
          } else {
              batchIndices.push_back(indices);
              numKept += numDetections;
        }
    }

    for (int i = 0 ; i < batchSize ; ++i) {
        int count = 0 ; // fixed size outputs
        int boxIdxOffset = numPriors * 4 * i ;
        int confIdxOffset = numClasses * numPriors * i ;

        for (std::map<int, std::vector<int> >::iterator it 
             = batchIndices[i].begin() ; it != batchIndices[i].end() ; ++it) {

            int label = it->first ;
            std::vector<int> &indices = it->second ;
            T* boxes_ = h_decodedBoxes + boxIdxOffset;
            T* confPreds_ = h_permutedConfPreds + confIdxOffset + label * numPriors ;

            int numIndices = indices.size() ;
            for (int j = 0 ; j < numIndices ; ++j) {
                int idx = indices[j] ;
                output[outHeight * i * 6 + count ] = label + 1 ; // MATLAB +1
                output[outHeight * i * 6 + outHeight + 1 * count] = confPreds_[idx] ;
                output[outHeight * i * 6 + outHeight * 2 + count] = boxes_[idx * 4] ;
                output[outHeight * i * 6 + outHeight * 3 + count] = boxes_[idx * 4 + 1] ;
                output[outHeight * i * 6 + outHeight * 4 + count] = boxes_[idx * 4 + 2] ;
                output[outHeight * i * 6 + outHeight * 5 + count] = boxes_[idx * 4 + 3] ;
              ++count;
            }
        }
    }
    delete[] h_decodedBoxes ;
    delete[] h_permutedConfPreds ;

    return VLE_Success ;
   }
 } ;
} } // namespace vl::impl

template struct vl::impl::multiboxdetector<vl::VLDT_GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::multiboxdetector<vl::VLDT_GPU, double> ;
#endif
